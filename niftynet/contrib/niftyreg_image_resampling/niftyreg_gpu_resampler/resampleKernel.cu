#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include"_reg_resampling.h"
#include"_reg_maths.h"
#include "resampleKernel.h"
#include "_reg_common_cuda.h"
#include"_reg_tools.h"

#define SINC_KERNEL_RADIUS 3
#define SINC_KERNEL_SIZE SINC_KERNEL_RADIUS*2

/* *************************************************************** */
unsigned int min1(unsigned int a, unsigned int b)
{
  return (a < b) ? a : b;
}
/* *************************************************************** */
template<class DTYPE>
__device__ __inline__ void reg_mat44_mul_cuda(DTYPE const* mat, DTYPE const* in, DTYPE *out)
{
  out[0] = (DTYPE)((double)mat[0 * 4 + 0] * (double)in[0] + (double)mat[0 * 4 + 1] * (double)in[1] + (double)mat[0 * 4 + 2] * (double)in[2] + (double)mat[0 * 4 + 3]);
  out[1] = (DTYPE)((double)mat[1 * 4 + 0] * (double)in[0] + (double)mat[1 * 4 + 1] * (double)in[1] + (double)mat[1 * 4 + 2] * (double)in[2] + (double)mat[1 * 4 + 3]);
  out[2] = (DTYPE)((double)mat[2 * 4 + 0] * (double)in[0] + (double)mat[2 * 4 + 1] * (double)in[1] + (double)mat[2 * 4 + 2] * (double)in[2] + (double)mat[2 * 4 + 3]);
  return;
}
/* *************************************************************** */
template<class DTYPE>
__device__ __inline__ void reg_mat44_mul_cuda(float* mat, DTYPE const* in, DTYPE *out)
{
  out[0] = (DTYPE)((double)mat[0 * 4 + 0] * (double)in[0] + (double)mat[0 * 4 + 1] * (double)in[1] + (double)mat[0 * 4 + 2] * (double)in[2] + (double)mat[0 * 4 + 3]);
  out[1] = (DTYPE)((double)mat[1 * 4 + 0] * (double)in[0] + (double)mat[1 * 4 + 1] * (double)in[1] + (double)mat[1 * 4 + 2] * (double)in[2] + (double)mat[1 * 4 + 3]);
  out[2] = (DTYPE)((double)mat[2 * 4 + 0] * (double)in[0] + (double)mat[2 * 4 + 1] * (double)in[1] + (double)mat[2 * 4 + 2] * (double)in[2] + (double)mat[2 * 4 + 3]);
  return;
}
/* *************************************************************** */
__device__ __inline__ int cuda_reg_floor(double a)
{
  return (int) (floor(a));
}
/* *************************************************************** */
template<class FieldTYPE>
__device__ __inline__ void interpolantCubicSpline(FieldTYPE ratio, FieldTYPE *basis)
{
  if (ratio < 0.0)
    ratio = 0.0; //reg_rounding error
  double FF = (double) ratio * ratio;
  basis[0] = (FieldTYPE) ((ratio * (((double)2.0 - ratio) * ratio - (double)1.0)) / (double)2.0);
  basis[1] = (FieldTYPE) ((FF * ((double)3.0 * ratio - 5.0) + 2.0) / (double)2.0);
  basis[2] = (FieldTYPE) ((ratio * (((double)4.0 - (double)3.0 * ratio) * ratio + (double)1.0)) / (double)2.0);
  basis[3] = (FieldTYPE) ((ratio - (double)1.0) * FF / (double)2.0);
}
/* *************************************************************** */
__inline__ __device__ void interpWindowedSincKernel(double relative, double *basis)
{
  if (relative < 0.0)
    relative = 0.0; //reg_rounding error
  int j = 0;
  double sum = 0.;
  for (int i = -SINC_KERNEL_RADIUS; i < SINC_KERNEL_RADIUS; ++i) {
    double x = relative - (double) (i);
    if (x == 0.0)
      basis[j] = 1.0;
    else if (abs(x) >= (double) (SINC_KERNEL_RADIUS))
      basis[j] = 0;
    else {
      double pi_x = M_PI * x;
      basis[j] = (SINC_KERNEL_RADIUS) * sin(pi_x) * sin(pi_x / SINC_KERNEL_RADIUS) / (pi_x * pi_x);
    }
    sum += basis[j];
    j++;
  }
  for (int i = 0; i < SINC_KERNEL_SIZE; ++i)
    basis[i] /= sum;
}
/* *************************************************************** */
__inline__ __device__ void interpCubicSplineKernel(double relative, double *basis)
{
  if (relative < 0.0)
    relative = 0.0; //reg_rounding error
  double FF = relative * relative;
  basis[0] = (relative * ((2.0 - relative) * relative - 1.0)) / 2.0;
  basis[1] = (FF * (3.0 * relative - 5.0) + 2.0) / 2.0;
  basis[2] = (relative * ((4.0 - 3.0 * relative) * relative + 1.0)) / 2.0;
  basis[3] = (relative - 1.0) * FF / 2.0;
}
/* *************************************************************** */
__inline__ __device__ void interpLinearKernel(double relative, double *basis)
{
  if (relative < 0.0)
    relative = 0.0; //reg_rounding error
  basis[1] = relative;
  basis[0] = 1.0 - relative;
}
/* *************************************************************** */
__inline__ __device__ void interpNearestNeighKernel(double relative, double *basis)
{
  if (relative < 0.0)
    relative = 0.0; //reg_rounding error
  basis[0] = basis[1] = 0.0;
  if (relative >= 0.5)
    basis[1] = 1;
  else
    basis[0] = 1;
}
/* *************************************************************** */
__inline__ __device__ double interpLoop2D(const float* floatingIntensity,
                                          double* xBasis,
                                          double* yBasis,
                                          double* zBasis,
                                          int *previous,
                                          uint3 fi_xyz,
                                          const float paddingValue,
                                          const unsigned int kernel_size)
{
  double intensity = (double)(0.0);

  for (int b = 0; b < kernel_size; b++) {
    int Y = previous[1] + b;
    bool yInBounds = -1 < Y && Y < fi_xyz.y;
    double xTempNewValue = 0.0;

    for (int a = 0; a < kernel_size; a++) {
      int X = previous[0] + a;
      bool xInBounds = -1 < X && X < fi_xyz.x;

      const unsigned int idx = Y * fi_xyz.x + X;

      xTempNewValue += (xInBounds && yInBounds) ? floatingIntensity[idx] * xBasis[a] : paddingValue * xBasis[a];
    }
    intensity += xTempNewValue * yBasis[b];
  }
  return intensity;
}
/* *************************************************************** */
__inline__ __device__ double interpLoop2DClamping(const float* floatingIntensity,
                                                  double* xBasis,
                                                  double* yBasis,
                                                  double* zBasis,
                                                  int *previous,
                                                  uint3 fi_xyz,
                                                  const unsigned int kernel_size)
{
  double intensity = (double)(0.0);

  for (int b = 0; b < kernel_size; b++) {
    const int offset_x = clampIndex(previous[1] + b, fi_xyz.y)*fi_xyz.x;

    double xTempNewValue = 0.0;

    for (int a = 0; a < kernel_size; a++) {
      const unsigned int idx = offset_x + clampIndex(previous[0] + a, fi_xyz.x);

      xTempNewValue += floatingIntensity[idx]*xBasis[a];
    }
    intensity += xTempNewValue*yBasis[b];
  }

  return intensity;
}
/* *************************************************************** */
__inline__ __device__ double interpLoop3D(const float* floatingIntensity,
                                          double* xBasis,
                                          double* yBasis,
                                          double* zBasis,
                                          int *previous,
                                          uint3 fi_xyz,
                                          float paddingValue,
                                          unsigned int kernel_size)
{
  double intensity = (double)(0.0);
  for (int c = 0; c < kernel_size; c++) {
    int Z = previous[2] + c;
    bool zInBounds = -1 < Z && Z < fi_xyz.z;
    double yTempNewValue = 0.0;
    for (int b = 0; b < kernel_size; b++) {
      int Y = previous[1] + b;
      bool yInBounds = -1 < Y && Y < fi_xyz.y;
      double xTempNewValue = 0.0;
      for (int a = 0; a < kernel_size; a++) {
        int X = previous[0] + a;
        bool xInBounds = -1 < X && X < fi_xyz.x;
        const unsigned int idx = Z * fi_xyz.x * fi_xyz.y + Y * fi_xyz.x + X;

        xTempNewValue += (xInBounds && yInBounds && zInBounds) ? floatingIntensity[idx] * xBasis[a] : paddingValue * xBasis[a];
      }
      yTempNewValue += xTempNewValue * yBasis[b];
    }
    intensity += yTempNewValue * zBasis[c];
  }
  return intensity;
}
/* *************************************************************** */
__inline__ __device__ double interpLoop3DClamping(const float* floatingIntensity,
                                                  double* xBasis,
                                                  double* yBasis,
                                                  double* zBasis,
                                                  int *previous,
                                                  uint3 fi_xyz,
                                                  unsigned int kernel_size)
{
  double intensity = (double)(0.0);
  for (int c = 0; c < kernel_size; c++) {
    const int offset_y = clampIndex(previous[2] + c, fi_xyz.z)*fi_xyz.y;

    double yTempNewValue = 0.0;

    for (int b = 0; b < kernel_size; b++) {
      const int offset_x = (offset_y + clampIndex(previous[1] + b, fi_xyz.y))*fi_xyz.x;

      double xTempNewValue = 0.0;

      for (int a = 0; a < kernel_size; a++) {
        const unsigned int idx = offset_x + clampIndex(previous[0] + a, fi_xyz.x);

        xTempNewValue += floatingIntensity[idx]*xBasis[a];
      }
      yTempNewValue += xTempNewValue*yBasis[b];
    }
    intensity += yTempNewValue*zBasis[c];
  }
  return intensity;
}
/* *************************************************************** */
template <const bool tDoClamp>
__global__ void ResampleImage2D(const float* floatingImage,
                                const float* deformationField,
                                float* warpedImage,
                                ulong2 voxelNumber,
                                uint3 fi_xyz,
                                uint2 wi_tu,
                                const float paddingValue,
                                const int kernelType)
{
  const float *sourceIntensityPtr = (floatingImage);
  float *resultIntensityPtr = (warpedImage);
  const float *deformationFieldPtrX = (deformationField);
  const float *deformationFieldPtrY = &deformationFieldPtrX[voxelNumber.x];

  long index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < voxelNumber.x) {

    for (unsigned int t = 0; t < wi_tu.x * wi_tu.y; t++) {

      float *resultIntensity = &resultIntensityPtr[t * voxelNumber.x];
      const float *floatingIntensity = &sourceIntensityPtr[t * voxelNumber.y];
      double intensity = paddingValue;

      int previous[3];
      float position[3];
      double relative[3];

      position[0] = (float)(deformationFieldPtrX[index]);
      position[1] = (float)(deformationFieldPtrY[index]);

      previous[0] = cuda_reg_floor(position[0]);
      previous[1] = cuda_reg_floor(position[1]);

      relative[0] = (double)(position[0]) - (double)(previous[0]);
      relative[1] = (double)(position[1]) - (double)(previous[1]);

      if (kernelType == 0) {

        double xBasisIn[2], yBasisIn[2], zBasisIn[2];
        interpNearestNeighKernel(relative[0], xBasisIn);
        interpNearestNeighKernel(relative[1], yBasisIn);
        if (tDoClamp) {
          intensity = interpLoop2DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 2);
        } else {
          intensity = interpLoop2D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 2);
        }
      }
      else if (kernelType == 1) {

        double xBasisIn[2], yBasisIn[2], zBasisIn[2];
        interpLinearKernel(relative[0], xBasisIn);
        interpLinearKernel(relative[1], yBasisIn);
        if (tDoClamp) {
          intensity = interpLoop2DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 2);
        } else {
          intensity = interpLoop2D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 2);
        }
      }
      else if (kernelType == 4) {

        double xBasisIn[6], yBasisIn[6], zBasisIn[6];

        previous[0] -= SINC_KERNEL_RADIUS;
        previous[1] -= SINC_KERNEL_RADIUS;
        previous[2] -= SINC_KERNEL_RADIUS;

        interpWindowedSincKernel(relative[0], xBasisIn);
        interpWindowedSincKernel(relative[1], yBasisIn);
        if (tDoClamp) {
          intensity = interpLoop2DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 6);
        } else {
          intensity = interpLoop2D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 6);
        }
      }
      else {

        double xBasisIn[4], yBasisIn[4], zBasisIn[4];

        previous[0]--;
        previous[1]--;
        previous[2]--;

        interpCubicSplineKernel(relative[0], xBasisIn);
        interpCubicSplineKernel(relative[1], yBasisIn);
        if (tDoClamp) {
          intensity = interpLoop2DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 4);
        } else {
          intensity = interpLoop2D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 4);
        }
      }

      resultIntensity[index] = (float)intensity;
    }
    index += blockDim.x * gridDim.x;
  }
}
/* *************************************************************** */
template <const bool tDoClamp>
__global__ void ResampleImage3D(const float* floatingImage,
                                const float* deformationField,
                                float* warpedImage,
                                const ulong2 voxelNumber,
                                uint3 fi_xyz,
                                uint2 wi_tu,
                                const float paddingValue,
                                int kernelType)
{
  const float *sourceIntensityPtr = (floatingImage);
  float *resultIntensityPtr = (warpedImage);
  const float *deformationFieldPtrX = (deformationField);
  const float *deformationFieldPtrY = &deformationFieldPtrX[voxelNumber.x];
  const float *deformationFieldPtrZ = &deformationFieldPtrY[voxelNumber.x];

  long index = blockIdx.x * blockDim.x + threadIdx.x;

  while (index < voxelNumber.x) {

    for (unsigned int t = 0; t < wi_tu.x * wi_tu.y; t++) {

      float *resultIntensity = &resultIntensityPtr[t * voxelNumber.x];
      const float *floatingIntensity = &sourceIntensityPtr[t * voxelNumber.y];
      double intensity = paddingValue;

      int previous[3];
      float position[3];
      double relative[3];

      position[0] = (float) (deformationFieldPtrX[index]);
      position[1] = (float) (deformationFieldPtrY[index]);
      position[2] = (float) (deformationFieldPtrZ[index]);

      previous[0] = cuda_reg_floor(position[0]);
      previous[1] = cuda_reg_floor(position[1]);
      previous[2] = cuda_reg_floor(position[2]);

      relative[0] = (double)(position[0]) - (double)(previous[0]);
      relative[1] = (double)(position[1]) - (double)(previous[1]);
      relative[2] = (double)(position[2]) - (double)(previous[2]);

      if (kernelType == 0) {

        double xBasisIn[2], yBasisIn[2], zBasisIn[2];
        interpNearestNeighKernel(relative[0], xBasisIn);
        interpNearestNeighKernel(relative[1], yBasisIn);
        interpNearestNeighKernel(relative[2], zBasisIn);
        if (tDoClamp) {
          intensity = interpLoop3DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 2);
        } else {
          intensity = interpLoop3D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 2);
        }
      } else if (kernelType == 1) {

        double xBasisIn[2], yBasisIn[2], zBasisIn[2];
        interpLinearKernel(relative[0], xBasisIn);
        interpLinearKernel(relative[1], yBasisIn);
        interpLinearKernel(relative[2], zBasisIn);
        if (tDoClamp) {
          intensity = interpLoop3DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 2);
        } else {
          intensity = interpLoop3D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 2);
        }
      } else if (kernelType == 4) {

        double xBasisIn[6], yBasisIn[6], zBasisIn[6];

        previous[0] -= SINC_KERNEL_RADIUS;
        previous[1] -= SINC_KERNEL_RADIUS;
        previous[2] -= SINC_KERNEL_RADIUS;

        interpWindowedSincKernel(relative[0], xBasisIn);
        interpWindowedSincKernel(relative[1], yBasisIn);
        interpWindowedSincKernel(relative[2], zBasisIn);
        if (tDoClamp) {
          intensity = interpLoop3DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 6);
        } else {
          intensity = interpLoop3D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 6);
        }
      } else {

        double xBasisIn[4], yBasisIn[4], zBasisIn[4];

        previous[0]--;
        previous[1]--;
        previous[2]--;

        interpCubicSplineKernel(relative[0], xBasisIn);
        interpCubicSplineKernel(relative[1], yBasisIn);
        interpCubicSplineKernel(relative[2], zBasisIn);
        if (tDoClamp) {
          intensity = interpLoop3DClamping(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, 4);
        } else {
          intensity = interpLoop3D(floatingIntensity, xBasisIn, yBasisIn, zBasisIn, previous, fi_xyz, paddingValue, 4);
        }
      }
      resultIntensity[index] = (float)intensity;
    }
    index += blockDim.x * gridDim.x;
  }
}
/* *************************************************************** */
void launchResample(const nifti_image *floatingImage,
                    const nifti_image *warpedImage,
                    const int interp,
                    const resampler_boundary_e boundary,
                    const float *floatingImage_d,
                    float *warpedImage_d,
                    const float *deformationFieldImage_d) {
  const float paddingValue = get_padding_value<float>(boundary);


  long targetVoxelNumber = (long) warpedImage->nx * warpedImage->ny * warpedImage->nz;
  ulong2 voxelNumber = make_ulong2(warpedImage->nx * warpedImage->ny * warpedImage->nz, floatingImage->nx * floatingImage->ny * floatingImage->nz);
  dim3 mygrid;
  dim3 myblocks;
  uint3 fi_xyz = make_uint3(floatingImage->nx, floatingImage->ny, floatingImage->nz);
  uint2 wi_tu = make_uint2(warpedImage->nt, warpedImage->nu);

  cudaCommon_computeGridConfiguration(myblocks, mygrid, targetVoxelNumber);
  if (floatingImage->nz > 1) {
    if (boundary == resampler_boundary_e::CLAMPING) {
      ResampleImage3D<true> <<<mygrid, myblocks >>>(floatingImage_d,
                                                    deformationFieldImage_d,
                                                    warpedImage_d,
                                                    voxelNumber,
                                                    fi_xyz,
                                                    wi_tu,
                                                    paddingValue,
                                                    interp);
    } else {
      ResampleImage3D<false> <<<mygrid, myblocks >>>(floatingImage_d,
                                                     deformationFieldImage_d,
                                                     warpedImage_d,
                                                     voxelNumber,
                                                     fi_xyz,
                                                     wi_tu,
                                                     paddingValue,
                                                     interp);
    }
  }
  else{
    if (boundary == resampler_boundary_e::CLAMPING) {
      ResampleImage2D<true> <<<mygrid, myblocks >>>(floatingImage_d,
                                                    deformationFieldImage_d,
                                                    warpedImage_d,
                                                    voxelNumber,
                                                    fi_xyz,
                                                    wi_tu,
                                                    paddingValue,
                                                    interp);
    } else {
      ResampleImage2D<false> <<<mygrid, myblocks >>>(floatingImage_d,
                                                     deformationFieldImage_d,
                                                     warpedImage_d,
                                                     voxelNumber,
                                                     fi_xyz,
                                                     wi_tu,
                                                     paddingValue,
                                                     interp);
    }
  }
#ifndef NDEBUG
  NR_CUDA_CHECK_KERNEL(mygrid, myblocks)
#else
    NR_CUDA_SAFE_CALL(hipDeviceSynchronize());
#endif
}
/* *************************************************************** */
__host__ nifti_image* resample(nifti_image &r_displacements, const nifti_image &floating, const int interp_code, const resampler_boundary_e boundary, const bool is_displacement_argument) {
  nifti_image *p_warped = nifti_copy_nim_info(&floating);

  assert(floating.dim[1] == r_displacements.dim[1] && floating.dim[2] == r_displacements.dim[2]
         && (floating.dim[3] == r_displacements.dim[3] || floating.ndim == 2));

  if (is_displacement_argument) {
    reg_getDeformationFromDisplacement(&r_displacements);
  }

  {
    float *dp_floating;
    float *dp_warped;
    float *dp_deformation;

    assert(floating.datatype == NIFTI_TYPE_FLOAT32);
    cudaCommon_allocateArrayToDevice(&dp_floating, floating.nvox);
    cudaCommon_allocateArrayToDevice(&dp_warped, floating.nvox);
    cudaCommon_allocateArrayToDevice(&dp_deformation, r_displacements.nvox);

    cudaCommon_transferNiftiToArrayOnDevice(&dp_floating, &floating);
    cudaCommon_transferNiftiToArrayOnDevice(&dp_deformation, &r_displacements);
    launchResample(&floating, p_warped, interp_code, boundary, dp_floating, dp_warped, dp_deformation);
    p_warped->data = std::malloc(p_warped->nvox*sizeof(float));
    cudaCommon_transferFromDeviceToNifti(p_warped, &dp_warped);

    hipFree(dp_floating);
    hipFree(dp_warped);
    hipFree(dp_deformation);
  }

  return p_warped;
}
/* *************************************************************** */
